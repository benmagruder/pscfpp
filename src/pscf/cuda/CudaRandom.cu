#include "CudaRandom.h"
#include "GpuTypes.h"

#include <util/global.h>

#include <hiprand.h>
#include <sys/time.h>

namespace Pscf {

   using namespace Util;

   /*
   * Constructor.
   */
   CudaRandom::CudaRandom()
    : gen_(),
      seed_(0),
      isInitialized_(false)
   {
      // Create pseudo-random number generator on gpu
      hiprandStatus_t status;
      status = hiprandCreateGenerator(&gen_, HIPRAND_RNG_PSEUDO_DEFAULT);
      UTIL_CHECK(status == HIPRAND_STATUS_SUCCESS);
   }

   /*
   * Destructor.
   */
   CudaRandom::~CudaRandom()
   {}

   /*
   * Sets of random seed, and initializes random number generator.
   *
   * \param seed value for random seed (private member variable seed)
   */
   void CudaRandom::setSeed(unsigned long long seed)
   {
      if (seed == 0) {
         timeval time;
         gettimeofday(&time, NULL);
         seed_ = time.tv_sec + 1123*time.tv_usec;
      } else {
         seed_ = seed;
      }
      hiprandStatus_t status;
      status = hiprandSetPseudoRandomGeneratorSeed(gen_, seed_);
      UTIL_CHECK(status == HIPRAND_STATUS_SUCCESS);
      isInitialized_ = true;
   }

   /*
   * Return uniformly distributed random number in [0,1]
   */
   double CudaRandom::uniform(cudaReal* data, int n)
   {
      if (!isInitialized_) {
         setSeed(0);
      }
      #ifdef SINGLE_PRECISION
      hiprandStatus_t gen_error 
           = hiprandGenerateUniform(gen_, data, int n);
      #else
      hiprandStatus_t gen_error 
            = hiprandGenerateUniformDouble(gen_, data, n);
      #endif
   }

}
