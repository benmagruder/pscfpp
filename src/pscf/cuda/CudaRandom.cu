#include "CudaRandom.h"
#include "GpuTypes.h"

#include <util/global.h>

#include <hiprand.h>
#include <sys/time.h>
#include <string>

namespace Pscf {

   using namespace Util;

   /*
   * Constructor.
   */
   CudaRandom::CudaRandom()
    : gen_(),
      seed_(0),
      isInitialized_(false)
   {
      // Create pseudo-random number generator on gpu
      hiprandStatus_t status;
      status = hiprandCreateGenerator(&gen_, HIPRAND_RNG_PSEUDO_DEFAULT);
      errorCheck(status);
   }

   /*
   * Destructor.
   */
   CudaRandom::~CudaRandom()
   {}

   /*
   * Sets of random seed, and initializes random number generator.
   *
   * \param seed value for random seed (private member variable seed)
   */
   void CudaRandom::setSeed(unsigned long long seed)
   {
      if (seed == 0) {
         timeval time;
         gettimeofday(&time, NULL);
         seed_ = time.tv_sec + 1123*time.tv_usec;
      } else {
         seed_ = seed;
      }
      hiprandStatus_t status;
      status = hiprandSetPseudoRandomGeneratorSeed(gen_, seed_);
      errorCheck(status);

      isInitialized_ = true;
   }

   /*
   * Return uniformly distributed random number in [0,1]
   */
   void CudaRandom::uniform(DeviceArray<cudaReal>& data)
   {
      UTIL_CHECK(data.capacity() > 0);
      if (!isInitialized_) {
         setSeed(0);
      }
      #ifdef SINGLE_PRECISION
      hiprandStatus_t status = hiprandGenerateUniform(gen_, data.cArray(), 
                                                    data.capacity());
      #else
      hiprandStatus_t status = hiprandGenerateUniformDouble(gen_, data.cArray(), 
                                                          data.capacity());
      #endif
      errorCheck(status);
   }

   /*
   * Return normal-distributed random floating point numbers.
   */
   void CudaRandom::normal(DeviceArray<cudaReal>& data, 
                           cudaReal stddev, cudaReal mean)
   {
      UTIL_CHECK(data.capacity() > 0);
      if (!isInitialized_) {
         setSeed(0);
      }

      int n = data.capacity();
      if (n % 2 == 1) {
         UTIL_THROW("normal() requires array size to be an even number.");
      }
      
      #ifdef SINGLE_PRECISION
      hiprandStatus_t status = hiprandGenerateNormal(gen_, data.cArray(), 
                                                   n, mean, stddev);
      #else
      hiprandStatus_t status = hiprandGenerateNormalDouble(gen_, data.cArray(), 
                                                         n, mean, stddev);
      #endif
      errorCheck(status);
   }

   /*
   * Check generator error status. If not success, print info and throw error.
   */
   void CudaRandom::errorCheck(hiprandStatus_t const & error)
   {
      if (error == HIPRAND_STATUS_SUCCESS) {
         return;
      } else {
         std::string errString;
         switch (error)
         {
            default:
               errString = "UNKNOWN";
               break;
            case HIPRAND_STATUS_VERSION_MISMATCH:
               errString = "HIPRAND_STATUS_VERSION_MISMATCH";
               break;
            case HIPRAND_STATUS_NOT_INITIALIZED:
               errString = "HIPRAND_STATUS_NOT_INITIALIZED";
               break;
            case HIPRAND_STATUS_ALLOCATION_FAILED:
               errString = "HIPRAND_STATUS_ALLOCATION_FAILED";
               break;
            case HIPRAND_STATUS_TYPE_ERROR:
               errString = "HIPRAND_STATUS_TYPE_ERROR";
               break;
            case HIPRAND_STATUS_OUT_OF_RANGE:
               errString = "HIPRAND_STATUS_OUT_OF_RANGE";
               break;
            case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:
               errString = "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";
               break;
            case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:
               errString = "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";
               break;
            case HIPRAND_STATUS_LAUNCH_FAILURE:
               errString = "HIPRAND_STATUS_LAUNCH_FAILURE";
               break;
            case HIPRAND_STATUS_PREEXISTING_FAILURE:
               errString = "HIPRAND_STATUS_PREEXISTING_FAILURE";
               break;
            case HIPRAND_STATUS_INITIALIZATION_FAILED:
               errString = "HIPRAND_STATUS_INITIALIZATION_FAILED";
               break;
            case HIPRAND_STATUS_INTERNAL_ERROR:
               errString = "HIPRAND_STATUS_INTERNAL_ERROR";
               break;
            case HIPRAND_STATUS_ARCH_MISMATCH:
               errString = "HIPRAND_STATUS_ARCH_MISMATCH";
               break;
         }

         Log::file() << "CudaRandom error: " << errString << std::endl;
         UTIL_THROW("CudaRandom number generation failed.");
      }
   }

}
