#include "hip/hip_runtime.h"
#ifndef PSCF_THREAD_ARRAY_CU
#define PSCF_THREAD_ARRAY_CU

#include "ThreadArray.h"
#include <hip/hip_runtime.h>

namespace {

   // Anonymous namespace containing "static" variables only used by global
   // functions defined in namespace ThreadArray. These are thus persistent
   // pseudo-private variables, much like private static class variables.

   // Maximum threads per block, either set by querying hardware or by user.
   int MAX_THREADS_PER_BLOCK = -1;

   // Number of threads per block for execution. 
   // Determined by setThreadsLogical.
   int THREADS_PER_BLOCK = -1;

   // Number of blocks for execution. Determined by setThreadsLogical.
   int BLOCKS = -1;

   // Total number of threads requested for execution. 
   // Set by setThreadsLogical.
   int THREADS_LOGICAL = -1;

   // Number of threads per warp
   int WARP_SIZE = -1;

   // Will threads go unused?
   bool UNUSED_THREADS;

}

namespace Pscf {
namespace ThreadArray {

   using namespace Util;

   void init()
   {
      // Check that a CUDA device is available.
      int count = 0;
      hipGetDeviceCount(&count);

      if (count == 0) {
         UTIL_THROW("No CUDA devices found.");
      } else if (count > 1) {
         Log::file() << "\nWarning: multiple GPUs detected.\n"
            << "This program is not compatible with multiple devices.\n"
            << "Only the first device will be used." << std::endl;
      }

      // Set a default maximum threads per block by querying hardware.
      setThreadsPerBlock();
   }

   void setThreadsPerBlock()
   {
      hipDeviceProp_t dprop;
      // Get properties, assuming one GPU.
      hipGetDeviceProperties(&dprop, 0);
      int maxThPerSM = dprop.maxThreadsPerMultiProcessor;

      // Find the highest power of two that evenly divides into the
      // maximum number of threads per streaming multiprocessor
      // This will lead to the highest occupancy!

      int threadsPerBlock = (maxThPerSM & (~(maxThPerSM - 1)));
      
      // Check for validity:
      while (threadsPerBlock > dprop.maxThreadsPerBlock) {
         threadsPerBlock /= 2;
      }

      setThreadsPerBlock(threadsPerBlock);
   }

   void setThreadsPerBlock(int nThreadsPerBlock)
   {
      MAX_THREADS_PER_BLOCK = nThreadsPerBlock;
      BLOCKS = 0;
      THREADS_LOGICAL = 0;
      checkExecutionConfig();
   }

   void setThreadsLogical(int nThreadsLogical)
   {
      // Verify that requested threads is valid (greater than 0).
      UTIL_ASSERT(nThreadsLogical > 0);
      
      // If max_threads_per_block hasn't been set at all, initialize.
      if (MAX_THREADS_PER_BLOCK == -1) 
         init();

      // Check if requested number of threads matches the previous request
      if (THREADS_LOGICAL == nThreadsLogical) {
         // Do nothing. Previous execution configuration will be used.
         return;
      }

      // Set the number of total requested threads.
      THREADS_LOGICAL = nThreadsLogical;

      // Compute the execution configuration. 
      // Number of blocks rounded up to the nearest integer.
      THREADS_PER_BLOCK = MAX_THREADS_PER_BLOCK;
      BLOCKS = ceil(double(nThreadsLogical)/double(THREADS_PER_BLOCK));

      // Determine if there will be unused threads
      UNUSED_THREADS = (BLOCKS*THREADS_PER_BLOCK > THREADS_LOGICAL);

   }

   void setThreadsLogical(int nThreadsLogical, int & nBlocks)
   {
      setThreadsLogical(nThreadsLogical);

      nBlocks = BLOCKS;
   }

   void 
   setThreadsLogical(int nThreadsLogical, int& nBlocks, int& nThreads)
   {
      setThreadsLogical(nThreadsLogical);

      nBlocks = BLOCKS;
      nThreads = THREADS_PER_BLOCK;
   }

   void checkExecutionConfig()
   {
      // Get relevant device hardware properties, assuming one device.
      hipDeviceProp_t dprop;
      hipGetDeviceProperties(&dprop, 0);
      WARP_SIZE = dprop.warpSize;
      int maxThreadsPerMultiProcessor = dprop.maxThreadsPerMultiProcessor;

      // Check that threads per block is a power of two. 
      // This is required for parallel reductions.
      if ((MAX_THREADS_PER_BLOCK & (MAX_THREADS_PER_BLOCK - 1)) != 0) {
         UTIL_THROW("Threads per block must be a power of two.");
      }

      // Check that threads per block is multiple of WARP_SIZE.
      // This is required because a warp is generally 32.
      if (MAX_THREADS_PER_BLOCK % WARP_SIZE != 0)
      {
         char buffer[100];
         sprintf(buffer, 
                 "Threads per block must be a multiple of warp size %d.\n",
                 WARP_SIZE);
         UTIL_THROW(buffer);
      }

      // Check that the maximum number of threads per multiprocessor is an 
      // integer multiple of the threads per block. This is not required 
      // for validity, but performance will be suboptimal if not the case, 
      // as it will limit the total number of threads that can be 
      // scheduled at any given time.

      if (maxThreadsPerMultiProcessor % MAX_THREADS_PER_BLOCK != 0) 
      {
         std::cerr 
              << "WARNING: The number of threads per block (" 
              << MAX_THREADS_PER_BLOCK 
              << ") is not an even divisor of the maximum number"
              << " of threads per streaming multiprocessor ("
              << maxThreadsPerMultiProcessor 
              << "). Performance will be suboptimal." 
              << std::endl;
      }

   }

   // Accessors

   int nBlocks()
   { return BLOCKS; }

   int nThreads()
   { return THREADS_PER_BLOCK; }

   int nThreadsLogical()
   { return THREADS_LOGICAL; }

   int warpSize()
   { return WARP_SIZE; }

   bool hasUnusedThreads()
   { return UNUSED_THREADS; }

}
}
#endif
