/*
* PSCF - Polymer Self-Consistent Field Theory
*
* Copyright 2015 - 2025, The Regents of the University of Minnesota
* Distributed under the terms of the GNU General Public License.
*/

#include "LrAmPreCompressor.tpp"

namespace Pscf {
namespace Rpg {

   template class LrAmPreCompressor<1>;
   template class LrAmPreCompressor<2>;
   template class LrAmPreCompressor<3>;

}
}
