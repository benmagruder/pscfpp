/*
* This program runs all unit tests in the rpg/tests/sweep directory.
*/ 

#include <util/global.h>
#include "SweepTestComposite.h"

#include <test/TestRunner.h>
#include <test/CompositeTestRunner.h>

int main(int argc, char* argv[])
{
   SweepTestComposite runner;
   runner.run();
}
