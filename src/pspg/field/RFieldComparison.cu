#define PSPG_R_FIELD_COMPARISON_CU

/*
* PSCF - Polymer Self-Consistent Field Theory
*
* Copyright 2016 - 2022, The Regents of the University of Minnesota
* Distributed under the terms of the GNU General Public License.
*/

#include "RFieldComparison.tpp"

namespace Pscf {
namespace Pspg
{

   template class RFieldComparison<1>;
   template class RFieldComparison<2>;
   template class RFieldComparison<3>;

} // namespace Pscf::Pspg
} // namespace Pscf
