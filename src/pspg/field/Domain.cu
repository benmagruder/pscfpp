/*
* PSCF - Polymer Self-Consistent Field Theory
*
* Copyright 2016 - 2019, The Regents of the University of Minnesota
* Distributed under the terms of the GNU General Public License.
*/

#include "Domain.tpp"

namespace Pscf {
namespace Pspg
{

   template class Domain<1>;
   template class Domain<2>;
   template class Domain<3>;

} // namespace Pspg
} // namespace Pscf
