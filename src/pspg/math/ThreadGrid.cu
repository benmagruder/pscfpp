#include "hip/hip_runtime.h"
#ifndef PSPG_THREADGRID_CU
#define PSPG_THREADGRID_CU

#include "ThreadGrid.h"

namespace {
   // Maximum threads per block, either set by querying hardware or by user.
   int MAX_THREADS_PER_BLOCK = -1;
   // Number of threads per block for execution. Determined by setThreadsLogical.
   int THREADS_PER_BLOCK = -1;
   // Number of blocks for execution. Determined by setThreadsLogical.
   int BLOCKS = -1;
   // Total number of threads requested for execution. Set by setThreadsLogical.
   int THREADS_LOGICAL = -1;
   // Will threads go unused?
   bool UNUSED_THREADS;
}

namespace Pscf {
namespace Pspg {
namespace ThreadGrid {

   using namespace Util;

   void init()
   {
      setThreadsPerBlock();

      // Check that a CUDA device is available.
      int count = 0;
      hipGetDeviceCount(&count);

      if (count == 0) 
         UTIL_THROW("No CUDA devices found.");
   }

   void setThreadsPerBlock()
   {
      hipDeviceProp_t dprop;
      // get properties, assuming one GPU.
      hipGetDeviceProperties(&dprop, 0);

      setThreadsPerBlock(dprop.maxThreadsPerBlock);

   }

   void setThreadsPerBlock(int const nThreadsPerBlock)
   {
      MAX_THREADS_PER_BLOCK = nThreadsPerBlock;
      BLOCKS = 0;
      THREADS_LOGICAL = 0;
      checkExecutionConfig();
   }

   void setThreadsLogical(int const nThreadsLogical)
   {
      if (MAX_THREADS_PER_BLOCK == -1) 
         UTIL_THROW("Number of threads per block not set before call to setThreadsLogical().");
      
      // Verify that requested threads is valid (greater than 0).
      UTIL_ASSERT(nThreadsLogical > 0);

      // Check if the number of requested threads matches the previous number of request threads
      if (THREADS_LOGICAL == nThreadsLogical) {
         // Do nothing. Previous execution configuration will be used.
         return;
      }

      // Set the number of total requested threads.
      THREADS_LOGICAL = nThreadsLogical;

      // Compute the execution configuration, with number of blocks rounded up to the nearest integer.
      THREADS_PER_BLOCK = MAX_THREADS_PER_BLOCK;
      BLOCKS = ceil((float)nThreadsLogical/MAX_THREADS_PER_BLOCK);

      // Determine if there will be unused threads
      if (BLOCKS*THREADS_PER_BLOCK > nThreadsLogical)
         UNUSED_THREADS = true;

   }

   void setThreadsLogical(int const nThreadsLogical, int & nBlocks)
   {
      setThreadsLogical(nThreadsLogical);

      nBlocks = BLOCKS;
   }

   void setThreadsLogical(int const nThreadsLogical, int & nBlocks, int & nThreads)
   {
      setThreadsLogical(nThreadsLogical);

      nBlocks = BLOCKS;
      nThreads = THREADS_PER_BLOCK;
   }

   void checkExecutionConfig()
   {
      // get relevant device hardware properties, assuming one device.
      hipDeviceProp_t dprop;
      hipGetDeviceProperties(&dprop, 0);
      int warpSize = dprop.warpSize;
      int maxThreadsPerMultiProcessor = dprop.maxThreadsPerMultiProcessor;

      // Check that threads per block is multiple of warpSize. This is required
      // because a warp is generally 32.
      if (MAX_THREADS_PER_BLOCK%warpSize != 0)
      {
         char buffer[100];
         sprintf(buffer, "Number of threads per block must be a multiple of warp size %d.\n",warpSize);
         UTIL_THROW(buffer);
      }

      // Check that the maximum number of threads per multiprocessor is an integer multiple
      // of the threads per block. This is not required for validity, but performance will be suboptimal
      // if not the case as it will limit the total number of threads that can be scheduled at any
      // given time.
      if (maxThreadsPerMultiProcessor % MAX_THREADS_PER_BLOCK%warpSize != 0)
      {
         std::cerr << "WARNING: The specified number of threads per block (" << MAX_THREADS_PER_BLOCK 
                  << ") is not an even divisor of the maximum number of threads per streaming multiprocessor ("
                  << maxThreadsPerMultiProcessor << "). Performance will be suboptimal." << std::endl;
      }

   }

   // Accessors

   int nBlocks()
   { return BLOCKS; }

   int nThreads()
   { return THREADS_PER_BLOCK; }

   int nThreadsLogical()
   { return THREADS_LOGICAL; }

   bool hasUnusedThreads()
   { return UNUSED_THREADS; }

}
}
}
#endif