#include "hip/hip_runtime.h"
#ifndef PSPG_LINEAR_ALGEBRA_CU
#define PSPG_LINEAR_ALGEBRA_CU

#include "LinearAlgebra.h"

namespace Pscf {
namespace Pspg {

__global__ void pointWiseAdd(cudaReal* result, const cudaReal* rhs, int size) 
{
   int nThreads = blockDim.x * gridDim.x;
   int startID = blockIdx.x * blockDim.x + threadIdx.x;
   for (int i = startID; i < size; i += nThreads) {
      result[i] += rhs[i];
   }
}

__global__ void subtractUniform(cudaReal* result, cudaReal rhs, int size) 
{
   int nThreads = blockDim.x * gridDim.x;
   int startID = blockIdx.x * blockDim.x + threadIdx.x;
   for (int i = startID; i < size; i += nThreads) {
      result[i] -= rhs;
   }
}

__global__ void addUniform(cudaReal* result, cudaReal rhs, int size) 
{
   int nThreads = blockDim.x * gridDim.x;
   int startID = blockIdx.x * blockDim.x + threadIdx.x;
   for (int i = startID; i < size; i += nThreads) {
      result[i] += rhs;
   }
}

__global__ void pointWiseSubtract(cudaReal* result, const cudaReal* rhs, int size) 
{
   int nThreads = blockDim.x * gridDim.x;
   int startID = blockIdx.x * blockDim.x + threadIdx.x;
   for (int i = startID; i < size; i += nThreads) {
      result[i] -= rhs[i];
   }
}

__global__ void pointWiseSubtractFloat(cudaReal* result, const float rhs, int size) 
{
   int nThreads = blockDim.x * gridDim.x;
   int startID = blockIdx.x * blockDim.x + threadIdx.x;
   for (int i = startID; i < size; i += nThreads) {
      result[i] -= rhs;
   }   
}

__global__ void pointWiseBinarySubtract(const cudaReal* a, const cudaReal* b, cudaReal* result, int size) 
{
   int nThreads = blockDim.x * gridDim.x;
   int startID = blockIdx.x * blockDim.x + threadIdx.x;
   for (int i = startID; i < size; i += nThreads) {
      result[i] = a[i] - b[i];
   }
}

__global__ void pointWiseBinaryAdd(const cudaReal* a, const cudaReal* b, cudaReal* result, int size) 
{
   int nThreads = blockDim.x * gridDim.x;
   int startID = blockIdx.x * blockDim.x + threadIdx.x;
   for (int i = startID; i < size; i += nThreads) {
      result[i] = a[i] + b[i];
   }
}

__global__ void pointWiseBinaryMultiply(const cudaReal* a, const cudaReal* b, cudaReal* result, int size) 
{
   int nThreads = blockDim.x * gridDim.x;
   int startID = blockIdx.x * blockDim.x + threadIdx.x;
   for (int i = startID; i < size; i += nThreads) {
      result[i] = a[i] * b[i];
   }
}

__global__ void pointWiseAddScale(cudaReal* result, const cudaReal* rhs, double scale, int size)
{
   int nThreads = blockDim.x * gridDim.x;
   int startID = blockIdx.x * blockDim.x + threadIdx.x;
   for (int i = startID; i < size; i += nThreads) {
      result[i] += scale * rhs[i];
   }
}

__global__ void assignUniformReal(cudaReal* result, cudaReal uniform, int size) {
   int nThreads = blockDim.x * gridDim.x;
   int startID = blockIdx.x * blockDim.x + threadIdx.x;
   for(int i = startID; i < size; i += nThreads) {
      result[i] = uniform;
   }
}

__global__ void assignReal(cudaReal* result, const cudaReal* rhs, int size) {
   int nThreads = blockDim.x * gridDim.x;
   int startID = blockIdx.x * blockDim.x + threadIdx.x;
   for(int i = startID; i < size; i += nThreads) {
      result[i] = rhs[i];
   }
}

__global__ void inPlacePointwiseMul(cudaReal* a, const cudaReal* b, int size) {
   int nThreads = blockDim.x * gridDim.x;
   int startID = blockIdx.x * blockDim.x + threadIdx.x;
   for(int i = startID; i < size; i += nThreads) {
      a[i] *= b[i];
   }
}

}
}
#endif
