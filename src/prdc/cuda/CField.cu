/*
* PSCF - Polymer Self-Consistent Field Theory
*
* Copyright 2016 - 2022, The Regents of the University of Minnesota
* Distributed under the terms of the GNU General Public License.
*/

#include "CField.tpp"

namespace Pscf {
namespace Prdc {
namespace Cuda {

   template class CField<1>;
   template class CField<2>;
   template class CField<3>;

} // namespace Pscf::Prdc::Cuda
} // namespace Pscf::Prdc
} // namespace Pscf
