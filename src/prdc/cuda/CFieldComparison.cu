/*
* PSCF - Polymer Self-Consistent Field Theory
*
* Copyright 2016 - 2022, The Regents of the University of Minnesota
* Distributed under the terms of the GNU General Public License.
*/

#include "CFieldComparison.tpp"

namespace Pscf {
namespace Prdc {
namespace Cuda {

   template class CFieldComparison<1>;
   template class CFieldComparison<2>;
   template class CFieldComparison<3>;

} // namespace Pscf::Prdc::Cuda
} // namespace Pscf::Prdc
} // namespace Pscf
